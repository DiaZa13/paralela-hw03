/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc vectorAdd.cu -o vectorAdd
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <fstream>


static const int BLOCK_SIZE = 256;
static const int N = 12000;

#define CUDA_CHECK_RETURN(value) {           \
    hipError_t _m_cudaStat = value;         \
    if (_m_cudaStat != hipSuccess) {        \
         fprintf(stderr, "Error %s at line %d in file %s\n",              \
                 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);    \
         exit(1);                                                         \
       } }

__global__ void vadd (const int *a, const int *b, int *c, int N)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
//  every thread just work on his id
    if (id < N)
        c[id] = a[id] + b[id];
}

int main (void)
{
//  host (h) and device (d) pointers
    int *ha, *hb, *hc, *da, *db, *dc;
    int i;

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

//  declare host vectors
    ha = new int[N];
    hb = new int[N];
    hc = new int[N];

//  declare device vectors
    CUDA_CHECK_RETURN (hipMalloc ((void **) &da, sizeof (int) * N)); //bloquea
    CUDA_CHECK_RETURN (hipMalloc ((void **) &db, sizeof (int) * N));
    CUDA_CHECK_RETURN (hipMalloc ((void **) &dc, sizeof (int) * N));

//  initialize host vectors
    for (i = 0; i < N; i++)
    {
        ha[i] = rand () % 10000;
        hb[i] = rand () % 10000;
    }

//  initialize device vectors
    CUDA_CHECK_RETURN (hipMemcpy (da, ha, sizeof (int) * N, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN (hipMemcpy (db, hb, sizeof (int) * N, hipMemcpyHostToDevice));

//  define grid size
    int grid = ceil (N * 1.0 / BLOCK_SIZE);
//  get initial time
    hipEventRecord(start, nullptr);
//  call kernel
    vadd <<< grid, BLOCK_SIZE >>> (da, db, dc, N);
//  synchronize
    CUDA_CHECK_RETURN (hipDeviceSynchronize ());
//  get end time
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
//  calculates the delta time
    hipEventElapsedTime(&time, start, stop);

    // Wait for the GPU launched work to complete
    CUDA_CHECK_RETURN (hipGetLastError ());
//  copy result vector from device to host
    CUDA_CHECK_RETURN (hipMemcpy (hc, dc, sizeof (int) * N, hipMemcpyDeviceToHost));

//  validate the sum is correct
    for (i = 0; i < N; i++)
    {
        if (hc[i] != ha[i] + hb[i])
            printf ("Error at index %i : %i VS %i\n", i, hc[i], ha[i] + hb[i]);
    }
    printf("Delta time: %f\n", time);
//  write result into file
    std::ofstream _file("../ex03/bitacora.txt", std::ios::app);
    _file << time << ",";
    _file.close();

//  free device vectors
    CUDA_CHECK_RETURN (hipFree ((void *) da));
    CUDA_CHECK_RETURN (hipFree ((void *) db));
    CUDA_CHECK_RETURN (hipFree ((void *) dc));
//  free host vector
    delete[]ha;
    delete[]hb;
    delete[]hc;

    //  destroy and clean resources
    CUDA_CHECK_RETURN (hipDeviceReset ());

    return 0;
}
