#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc vectorAdd.cu -o vectorAdd
 ============================================================================
 */
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>

static const int BLOCK_SIZE = 256;

#define CUDA_CHECK_RETURN(value) {           \
    hipError_t _m_cudaStat = value;         \
    if (_m_cudaStat != hipSuccess) {        \
         fprintf(stderr, "Error %s at line %d in file %s\n",              \
                 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);    \
         exit(1);                                                         \
       } }

//__global__ void dynamic_add(int *a, const int *b, const int n){
//    extern __shared__ int c[];
//    int id = blockIdx.x * blockDim.x + threadIdx.x;
//    int tid = threadIdx.x;
//    if (id < n){
//        c[tid] = a[id] + b[id];
//        __syncthreads();
//        a[id] = c[tid];
//    }
//}
__global__ void dynamic_add(int *a, const int *b, const int n){
    extern __shared__ int c[];
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    if (id < n){
        c[tid] = a[id] + b[id];
        __syncthreads();
        a[id] = c[tid];
    }

int main (void)
{
//  host (h) and device (d) pointers
    int *ha, *hb, *hc, *da, *db;
    int i, vector_size;

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    std::cout<<"Enter the vector size: ";
    std::cin>>vector_size;

//  declare host vectors
    ha = new int[vector_size];
    hb = new int[vector_size];
    hc = new int[vector_size];

//  declare device vectors
    CUDA_CHECK_RETURN (hipMalloc ((void **) &da, sizeof (int) * vector_size));
    CUDA_CHECK_RETURN (hipMalloc ((void **) &db, sizeof (int) * vector_size));

//  initialize host vectors
    for (i = 0; i < vector_size; i++)
    {
        ha[i] = rand () % 10000;
        hb[i] = rand () % 10000;
    }

//  initialize device vectors
    CUDA_CHECK_RETURN (hipMemcpy (da, ha, sizeof (int) * vector_size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN (hipMemcpy (db, hb, sizeof (int) * vector_size, hipMemcpyHostToDevice));


//  define grid size
    int grid = ceil (vector_size * 1.0 / BLOCK_SIZE);
//  get initial time
    hipEventRecord(start, nullptr);
//  allowing to use the 64KB of shared memory
//    hipFuncSetAttribute(reinterpret_cast<const void*>(dynamic_add), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
//  call kernel
    dynamic_add <<< grid, BLOCK_SIZE, sizeof (int) * BLOCK_SIZE>>> (da, db, vector_size);
//  synchronize
    CUDA_CHECK_RETURN (hipDeviceSynchronize ());
//  get end time
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
//  calculates the delta time
    hipEventElapsedTime(&time, start, stop);

//  Wait for the GPU launched work to complete
    CUDA_CHECK_RETURN (hipGetLastError ());
//  copy result vector from device to host
    CUDA_CHECK_RETURN (hipMemcpy (hc, da, sizeof (int) * vector_size, hipMemcpyDeviceToHost));

//  validate the sum is correct
    for (i = 0; i < vector_size; i++)
    {
        if (hc[i] != ha[i] + hb[i]) {
            printf ("Error at index %i : %i VS %i\n", i, hc[i], ha[i] + hb[i]);
        }
    }
    //  write result into file
    printf("Delta time: %f\n", time);
    std::ofstream _file("../ex03/bitacora.txt", std::ios::app);
    _file << time << ",";
    _file.close();

//  free device vectors
    CUDA_CHECK_RETURN (hipFree ((void *) da));
    CUDA_CHECK_RETURN (hipFree ((void *) db));
//  free host vector
    delete[]ha;
    delete[]hb;
    delete[]hc;

//  destroy and clean resources
    CUDA_CHECK_RETURN (hipDeviceReset ());

    return 0;
}
