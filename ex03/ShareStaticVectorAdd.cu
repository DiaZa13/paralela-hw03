#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc vectorAdd.cu -o vectorAdd
 ============================================================================
 */
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <fstream>


static const int BLOCK_SIZE = 256;
static const int N = 100000;

#define CUDA_CHECK_RETURN(value) {           \
    hipError_t _m_cudaStat = value;         \
    if (_m_cudaStat != hipSuccess) {        \
         fprintf(stderr, "Error %s at line %d in file %s\n",              \
                 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);    \
         exit(1);                                                         \
       } }

//__global__ void static_add(int *a, const int *b, const int n){
//    __shared__ int c[N];
//    int id = blockIdx.x * blockDim.x + threadIdx.x;
//    if (id < n){
//        c[id] = a[id] + b[id];
//        __syncthreads();
//        a[id] = c[id];
//    }
//}

__global__ void static_add(int *a, const int *b, const int n){
    __shared__ int c[BLOCK_SIZE];
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    if (id < n){
        c[tid] = a[id] + b[id];
        __syncthreads();
        a[id] = c[tid];
    }

int main (void)
{
//  host (h) and device (d) pointers
    int *ha, *hb, *hc, *da, *db;
    int i;
    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

//  declare host vectors
    ha = new int[N];
    hb = new int[N];
    hc = new int[N];

//  declare device vectors
    CUDA_CHECK_RETURN (hipMalloc ((void **) &da, sizeof (int) * N)); //bloquea
    CUDA_CHECK_RETURN (hipMalloc ((void **) &db, sizeof (int) * N));

//  initialize host vectors
    for (i = 0; i < N; i++)
    {
        ha[i] = rand () % 10000;
        hb[i] = rand () % 10000;
    }

//  initialize device vectors
    CUDA_CHECK_RETURN (hipMemcpy (da, ha, sizeof (int) * N, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN (hipMemcpy (db, hb, sizeof (int) * N, hipMemcpyHostToDevice));


//  define grid size
    int grid = ceil (N * 1.0 / BLOCK_SIZE);
//  get initial time
    hipEventRecord(start, nullptr);
//  call kernel
    static_add <<< grid, BLOCK_SIZE >>> (da, db, N);
//  synchronize
    CUDA_CHECK_RETURN (hipDeviceSynchronize ());
//  get end time
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
//  calculates the delta time
    hipEventElapsedTime(&time, start, stop);

    // Wait for the GPU launched work to complete
    CUDA_CHECK_RETURN (hipGetLastError ());
//  copy result vector from device to host
    CUDA_CHECK_RETURN (hipMemcpy (hc, da, sizeof (int) * N, hipMemcpyDeviceToHost));

//  validate the sum is correct
    for (i = 0; i < N; i++)
    {
        if (hc[i] != ha[i] + hb[i]) {
            printf ("Error at index %i : %i VS %i\n", i, hc[i], ha[i] + hb[i]);
        }
    }
    //  write result into file
    printf("Delta time: %f\n", time);
    std::ofstream _file("../ex03/bitacora.txt", std::ios::app);
    _file << time << ",";
    _file.close();

//  free device vectors
    CUDA_CHECK_RETURN (hipFree ((void *) da));
    CUDA_CHECK_RETURN (hipFree ((void *) db));
//  free host vector
    delete[]ha;
    delete[]hb;
    delete[]hc;

//  destroy and clean resources
    CUDA_CHECK_RETURN (hipDeviceReset ());

    return 0;
}
