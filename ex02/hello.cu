/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello2.cu -o hello2 -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello() {
//   int myID = ( blockIdx.z * gridDim.x * gridDim.y  +
//                blockIdx.y * gridDim.x +
//                blockIdx.x ) * blockDim.x * blockDim.y * blockDim.z +
//                threadIdx.z *  blockDim.x * blockDim.y +
//                threadIdx.y * blockDim.x +
//                threadIdx.x;

//  Simplificando...
    //grid: 3D --- z,y,x: all dims and blockids
    //block: 1D -- x
    int myID = (blockIdx.y * gridDim.x + blockIdx.x) *
               blockDim.x * blockDim.y + threadIdx.y *
                                         blockDim.x + threadIdx.x;

    printf("Diana Zaray Corado #191025 %i\n", myID);
}

int main() {
    dim3 g(4, 2);
    dim3 b(32, 16);
    hello <<< g, b >>>();
//    cudaThreadSynchronize ();
    hipDeviceSynchronize();  //use instead, ^ is deprecated
    return 0;
}
